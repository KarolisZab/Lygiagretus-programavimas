#include "hip/hip_runtime.h"
﻿#define __HIPCC__
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <fstream>
#include <nlohmann/json.hpp>

using namespace std;
using json = nlohmann::json;

class Item
{
public:
    char itemTitle[256];
    int itemCount;
    double itemCost;
    char res[256];
    int fp_sum;
};

const string dataFile = "IFF_0_6_ŽabinskisK_L1_dat_1.json"; // visi tinka
//const string dataFile = "IFF_0_6_ŽabinskisK_L1_dat_2.json"; // kaikurie tinka
//const string dataFile = "IFF_0_6_ŽabinskisK_L1_dat_3.json"; // nei vienas netinka
const string resultFile = "IFF_0_6_ŽabinskisK_L3_rez.txt";

void readItemsFiles(vector<Item>* items) {
    ifstream stream(dataFile);
    json allItemsJson = json::parse(stream);

    auto allItems = allItemsJson["items"];
    for (auto& new_items : allItems) {
        Item tempItem;
        string n = new_items["title"];
        //Returns a pointer to an array that contains a null-terminated sequence of characters (i.e., a C-string) 
        // representing the current value of the string object.
        strcpy(tempItem.itemTitle, n.c_str());
        tempItem.itemCount = new_items["count"];
        tempItem.itemCost = new_items["cost"];
        items->push_back(tempItem);
    }
    stream.close();
}

void writeListToFile(vector<Item>& items, string fileName) {
    ofstream file;
    file.open(fileName, ios::out);
    file << setw(33) << "Pradiniai duomenys" << endl
        << "--------------------------------------------------------------" << endl
        << setw(5) << "Nr. |" << setw(25) << "Pavadinimas |" << setw(15) << "Kiekis |" << setw(17) << "Kaina |" << endl
        << "--------------------------------------------------------------" << endl;

    for (int i = 0; i < items.size(); i++)
    {
        file << setw(5) << to_string(i) << setw(23) << items[i].itemTitle << " |" << setw(13) << to_string(items[i].itemCount) << " |" << setw(15) << to_string(items[i].itemCost) << " |" << endl;
    }
    file << "--------------------------------------------------------------" << endl << endl;
    file.close();
}

void writeResultToFile(Item items[], string fileName, int res_size) {
    ofstream file;
    file.open(fileName, ios::app);
    file << setw(39) << "Rezultatai" << endl
        << "---------------------------------------------------------------------------------" << endl
        << setw(5) << "Nr. |" << setw(25) << "Pavadinimas |" << setw(15) << "Kiekis |" << setw(17) << "Kaina |" << setw(16) << "Teksto rezultatas |" << endl
        << "---------------------------------------------------------------------------------" << endl;

    for (int i = 0; i < res_size; i++)
    {
        file << setw(5) << to_string(i) << setw(23) << items[i].itemTitle << " |" << setw(13) << to_string(items[i].itemCount) << " |"
            << setw(15) << to_string(items[i].itemCost) << " |" << setw(17) << (items[i].res) << " |" << setw(20) << to_string(items[i].fp_sum) << "|" << endl;
    }
    file << "---------------------------------------------------------------------------------" << endl;
    file << "Total Price:"  << endl;
    file.close();
}

//__device__  Funkcijos, vykdomos GPU ir kviečiamos iš GPU

// Appends char array to other char array
__device__ void gpu_strcpy(char* dest, const char* src) {
    int i = 0;
    do {
        dest[i] = src[i];
    } while (src[i++] != 0);
}

__device__ void gpu_string(char* dest, const char* src) {
    dest[0] = src[0];
    dest[1] = src[1];
    dest[2] = '<';
    dest[3] = '2';
    dest[4] = '0';
    dest[5] = '0';
}

//__global__ Funkcijos, vykdomos GPU, bet kviečiamos iš CPU
__global__ void gpu_func(Item* device_items, Item* device_results, int* device_array_size, int* device_slice_size, int* device_result_count) {
    
    // compute start index
    unsigned long start_index = *device_slice_size * threadIdx.x;
    unsigned long end_index;


    // compute end index. Last thread takes all remaining elements in case they are not split evenly between threads
    if (threadIdx.x == blockDim.x - 1)
        end_index = *device_array_size;
    else
        end_index = *device_slice_size * (threadIdx.x + 1);

    auto fp_sum = 0;

    for (int i = start_index; i < end_index; i++) {
        double quantity = device_items[i].itemCount;
        double price = device_items[i].itemCost;
        double fullPrice = quantity * price;

        if (fullPrice <= 200) {
            Item Item;
            gpu_strcpy(Item.itemTitle, device_items[i].itemTitle);
            Item.itemCount = device_items[i].itemCount;
            Item.itemCost = device_items[i].itemCost;
            gpu_string(Item.res, device_items[i].itemTitle);
            Item.fp_sum += device_items[i].itemCost * device_items[i].itemCount;
            // sudeda dvi reiksmes
            int index = atomicAdd(device_result_count, 1);
            device_results[index] = Item;
        }
    }
}

const int SIZE = 256;
// Intel(R) Core(TM) i5-8300H CPU - turi 4 bandruolius
// Palaiko 8 gijas
const int THREADS = 8;

int main()
{
    vector<Item> data;
    readItemsFiles(&data);

    Item* items = &data[0];
    Item results[SIZE];
    // one thread processes one slice of data. Slice size is equal to total count divided by number of threads
    int slice_size = SIZE / THREADS;
    int result_count = 0;

    Item* device_items;
    Item* device_results;
    int* device_array_size;
    int* device_slice_size;
    int* device_result_count;


    //GPU atmintyje išskiria nurodytą kiekį atminties. 
    hipMalloc((void**)&device_items, SIZE * sizeof(Item));
    hipMalloc((void**)&device_array_size, sizeof(int));
    hipMalloc((void**)&device_slice_size, sizeof(int));
    hipMalloc((void**)&device_result_count, sizeof(int));
    hipMalloc((void**)&device_results, SIZE * sizeof(Item));

    //Funkcijos, vykdomos GPU ir kviečiamos iš GPU
    //hipMemcpyHostToHost iš CPU į CPU
    //hipMemcpyHostToDevice iš CPU į GPU
    //hipMemcpyDeviceToHost iš GPU į CPU
    //hipMemcpyDeviceToDevice iš GPU į GPU

    //Iš CPU siunčiami(kopijuojami) duomenys į GPU
    hipMemcpy(device_items, items, SIZE * sizeof(Item), hipMemcpyHostToDevice);
    hipMemcpy(device_array_size, &SIZE, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_slice_size, &slice_size, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_result_count, &result_count, sizeof(int), hipMemcpyHostToDevice);

    gpu_func << <1, THREADS >> > (device_items, device_results, device_array_size, device_slice_size, device_result_count);
    //Blokuoja CPU kodą, kol GPU pabaigs visą jam priskirtą darbą.
    hipDeviceSynchronize();

    hipMemcpy(&results, device_results, SIZE * sizeof(Item), hipMemcpyDeviceToHost);
    int RES_SIZE = 0;
    hipMemcpy(&RES_SIZE, device_result_count, sizeof(int), hipMemcpyDeviceToHost);

    writeListToFile(data, resultFile);
    writeResultToFile(results, resultFile, RES_SIZE);


    //Atlaisvina GPU išskirtą atmintį. 
    hipFree(device_array_size);
    hipFree(device_items);
    hipFree(device_results);
    hipFree(device_result_count);
    hipFree(device_slice_size);

    return 0;
}

